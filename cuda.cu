#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"
#include "test.h"

#define THREADS_PER_BLOCK 256


__global__ void forwardPropagationKernel(const int num_nodes, const int *row_ptr, const int *col_idx, int *fwd_label, const int *active, int *changed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_nodes || !active[tid]){
        return;
    }

    for (int i = row_ptr[tid]; i < row_ptr[tid+1]; i++) {
        int neighbour = col_idx[i];
        if (active[neighbour]) {
            int newVal = fwd_label[neighbour];
            int prev = atomicMin(&fwd_label[tid], newVal);
            if (newVal < prev) {
                atomicExch(changed, 1);
            }
        }
    }
}


__global__ void backwardPropagationKernel(const int num_nodes, const int *t_row_ptr, const int *t_col_idx, int *bwd_label, const int *active, const int *fwd_label, int *changed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_nodes || !active[tid]){
        return;
    }

    for (int i = t_row_ptr[tid]; i < t_row_ptr[tid+1]; i++) {
        int neighbour = t_col_idx[i];
        if (active[neighbour] && (fwd_label[tid] == fwd_label[neighbour])) {
            int newVal = bwd_label[neighbour];
            int prev = atomicMin(&bwd_label[tid], newVal);
            if (newVal < prev) {
                atomicExch(changed, 1);
            }
        }
    }
}


__global__ void resetLabelsKernel(int num_nodes, int *fwd_label, int *bwd_label, const int *active) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_nodes && active[tid]) {
        fwd_label[tid] = tid;
        bwd_label[tid] = tid;
    }
}


__global__ void intersectionKernel(const int num_nodes, const int *fwd_label, const int *bwd_label, int *active, int *scc) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_nodes || !active[tid]) {
        return;
    }

    if (fwd_label[tid] == bwd_label[tid]) {
        scc[tid] = fwd_label[tid];
        active[tid] = 0;
    }
}


int *parallel_cuda(CSRGraph *graph) {
    CSRGraph *tGraph = transposeCSRGraph(graph);

    int *d_row_ptr, *d_col_idx, *d_t_row_ptr, *d_t_col_idx;
    hipMalloc(&d_row_ptr, (graph -> num_nodes + 1) * sizeof(int));
    hipMalloc(&d_col_idx, graph -> num_edges * sizeof(int));
    hipMemcpy(d_row_ptr, graph -> row_ptr, (graph -> num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, graph -> col_idx, graph -> num_edges * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_t_row_ptr, (tGraph -> num_nodes + 1) * sizeof(int));
    hipMalloc(&d_t_col_idx, tGraph -> num_edges * sizeof(int));
    hipMemcpy(d_t_row_ptr, tGraph -> row_ptr, (tGraph -> num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_t_col_idx, tGraph -> col_idx, tGraph -> num_edges * sizeof(int), hipMemcpyHostToDevice);

    int *h_scc = (int*)malloc(graph -> num_nodes * sizeof(int));

    int *h_active = (int*)malloc(graph -> num_nodes * sizeof(int));
    for (int i = 0; i < graph -> num_nodes; i++){
        h_active[i] = 1;
    }

    int *d_fwd_label, *d_bwd_label, *d_active, *d_scc;
    hipMalloc(&d_fwd_label, graph -> num_nodes * sizeof(int));
    hipMalloc(&d_bwd_label, graph -> num_nodes * sizeof(int));
    hipMalloc(&d_active, graph -> num_nodes * sizeof(int));
    hipMalloc(&d_scc, graph -> num_nodes * sizeof(int));

    int *h_init = (int*)malloc(graph -> num_nodes * sizeof(int));
    for (int i = 0; i < graph -> num_nodes; i++){
        h_init[i] = i;
    }

    hipMemcpy(d_fwd_label, h_init, graph -> num_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bwd_label, h_init, graph -> num_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_active, h_active, graph -> num_nodes * sizeof(int), hipMemcpyHostToDevice);
    free(h_init);

    int blocks = (graph -> num_nodes + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int h_changed;
    int *d_changed;
    hipMalloc(&d_changed, sizeof(int));

    bool done = false;
    while (!done) {
        h_changed = 0;
        hipMemcpy(d_changed, &h_changed, sizeof(int), hipMemcpyHostToDevice);

        // forward process
        do {
            h_changed = 0;
            hipMemcpy(d_changed, &h_changed, sizeof(int), hipMemcpyHostToDevice);

            forwardPropagationKernel<<<blocks, THREADS_PER_BLOCK>>>(graph -> num_nodes, d_row_ptr, d_col_idx, d_fwd_label, d_active, d_changed);

            hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
        } while (h_changed);

        // backward process
        do {
            h_changed = 0;
            hipMemcpy(d_changed, &h_changed, sizeof(int), hipMemcpyHostToDevice);

            backwardPropagationKernel<<<blocks, THREADS_PER_BLOCK>>>(graph -> num_nodes, d_t_row_ptr, d_t_col_idx, d_bwd_label, d_active, d_fwd_label, d_changed);

            hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
        } while (h_changed);

        intersectionKernel<<<blocks, THREADS_PER_BLOCK>>>(graph -> num_nodes, d_fwd_label, d_bwd_label, d_active, d_scc);

        resetLabelsKernel<<<blocks, THREADS_PER_BLOCK>>>(graph -> num_nodes, d_fwd_label, d_bwd_label, d_active);

        int *h_temp = (int*)malloc(graph -> num_nodes * sizeof(int));
        hipMemcpy(h_temp, d_active, graph -> num_nodes * sizeof(int), hipMemcpyDeviceToHost);

        int activeCount = 0;
        for (int i = 0; i < graph -> num_nodes; i++) {
            activeCount += h_temp[i];
        }
        free(h_temp);

        if (activeCount == 0) {
            done = true;
        }
    }

    hipMemcpy(h_scc, d_scc, graph -> num_nodes * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(d_fwd_label);
    hipFree(d_bwd_label);
    hipFree(d_active);
    hipFree(d_scc);
    hipFree(d_changed);
    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_t_row_ptr);
    hipFree(d_t_col_idx);
    free(h_active);
    freeCSRGraph(tGraph);

    return h_scc;
}


void test_parallel_cuda(CSRGraph *graph) {
    int *scc = parallel_cuda(graph);

    printf("Node -> SCC\n");
    for (int i = 0; i < graph -> num_nodes; i++) {
        printf("%d -> %d\n", i, scc[i]);
    }

    free(scc);
}
